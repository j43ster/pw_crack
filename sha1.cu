#include "hip/hip_runtime.h"
/* This code is public-domain - it is based on libcrypt 
 * placed in the public domain by Wei Dai and other contributors.
 */
// gcc -Wall -DSHA1TEST -o sha1test sha1.c && ./sha1test

#include "sha1.h"

#define BLOCKS 512
#define MAX_BLOCKS 64000
#define HANDLE_ERROR( err ) (handle_error( err, __FILE__, __LINE__ ))

__device__ const uint8_t sha1InitState[] = {
  0x01,0x23,0x45,0x67, // H0
  0x89,0xab,0xcd,0xef, // H1
  0xfe,0xdc,0xba,0x98, // H2
  0x76,0x54,0x32,0x10, // H3
  0xf0,0xe1,0xd2,0xc3  // H4
};

// ----------------------------------------------------------------------------
//                    GPU FUNCTIONS
// ----------------------------------------------------------------------------

__device__ void d_sha1_init(sha1nfo *s) {
  memcpy(s->state.b,sha1InitState,HASH_LENGTH);
  s->byteCount = 0;
  s->bufferOffset = 0;
}

__device__ uint32_t d_sha1_rol32(uint32_t number, uint8_t bits) {
  return ((number << bits) | (number >> (32-bits)));
}

__device__ void d_sha1_hashBlock(sha1nfo *s) {
  uint8_t i;
  uint32_t a,b,c,d,e,t;

  a=s->state.w[0];
  b=s->state.w[1];
  c=s->state.w[2];
  d=s->state.w[3];
  e=s->state.w[4];
  for (i=0; i<80; i++) {
    if (i>=16) {
      t = s->buffer.w[(i+13)&15] ^ s->buffer.w[(i+8)&15] ^ s->buffer.w[(i+2)&15] ^ s->buffer.w[i&15];
      s->buffer.w[i&15] = d_sha1_rol32(t,1);
    }
    if (i<20) {
      t = (d ^ (b & (c ^ d))) + SHA1_K0;
    } else if (i<40) {
      t = (b ^ c ^ d) + SHA1_K20;
    } else if (i<60) {
      t = ((b & c) | (d & (b | c))) + SHA1_K40;
    } else {
      t = (b ^ c ^ d) + SHA1_K60;
    }
    t+=d_sha1_rol32(a,5) + e + s->buffer.w[i&15];
    e=d;
    d=c;
    c=d_sha1_rol32(b,30);
    b=a;
    a=t;
  }
  s->state.w[0] += a;
  s->state.w[1] += b;
  s->state.w[2] += c;
  s->state.w[3] += d;
  s->state.w[4] += e;
}

__device__ void d_sha1_addUncounted(sha1nfo *s, uint8_t data) {
  s->buffer.b[s->bufferOffset ^ 3] = data;
  s->bufferOffset++;
  if (s->bufferOffset == BLOCK_LENGTH) {
    d_sha1_hashBlock(s);
    s->bufferOffset = 0;
  }
}

__device__ void d_sha1_writebyte(sha1nfo *s, uint8_t data) {
  ++s->byteCount;
  d_sha1_addUncounted(s, data);
}

__device__ void d_sha1_write(sha1nfo *s, const char *data, size_t len) {
	for (;len--;) d_sha1_writebyte(s, (uint8_t) *data++);
}

__device__ void d_sha1_pad(sha1nfo *s) {
  // Implement SHA-1 padding (fips180-2 §5.1.1)

  // Pad with 0x80 followed by 0x00 until the end of the block
  d_sha1_addUncounted(s, 0x80);
  while (s->bufferOffset != 56) d_sha1_addUncounted(s, 0x00);

  // Append length in the last 8 bytes
  d_sha1_addUncounted(s, 0); // We're only using 32 bit lengths
  d_sha1_addUncounted(s, 0); // But SHA-1 supports 64 bit lengths
  d_sha1_addUncounted(s, 0); // So zero pad the top bits
  d_sha1_addUncounted(s, s->byteCount >> 29); // Shifting to multiply by 8
  d_sha1_addUncounted(s, s->byteCount >> 21); // as SHA-1 supports bitstreams as well as
  d_sha1_addUncounted(s, s->byteCount >> 13); // byte.
  d_sha1_addUncounted(s, s->byteCount >> 5);
  d_sha1_addUncounted(s, s->byteCount << 3);
}

__device__ uint8_t* d_sha1_result(sha1nfo *s) {
  int i;
  // Pad to complete the last block
  d_sha1_pad(s);
  
  // Swap byte order back
  for (i=0; i<5; i++) {
    uint32_t a,b;
    a=s->state.w[i];
    b=a<<24;
    b|=(a<<8) & 0x00ff0000;
    b|=(a>>8) & 0x0000ff00;
    b|=a>>24;
    s->state.w[i]=b;
  }
  
  // Return pointer to hash (20 characters)
  return s->state.b;
}

__global__ void crack_password (uint8_t *hash, char *password, int max_len) {

}

// ----------------------------------------------------------------------------
//                    CPU FUNCTIONS
// ----------------------------------------------------------------------------

void handle_error(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString( err ), file, line);
          exit(EXIT_FAILURE);
  }
}

extern "C" void run_kernel (uint8_t *hash, char *password, int max_len) {
  uint8_t *d_hash;
  char *d_password;
  int block_size, grid_size;

  // Setup Device Variables
  HANDLE_ERROR (hipMalloc (&d_hash, BLOCK_LENGTH));
  HANDLE_ERROR (hipMemcpy (d_hash, hash, BLOCK_LENGTH, hipMemcpyHostToDevice));

  HANDLE_ERROR (hipMalloc (&d_password, max_len));
  
  // Compute correct block and grid sizes
  block_size = BLOCKS;
  grid_size = MAX_BLOCKS;

  // Execute Kernel
  crack_password<<<block_size, grid_size>>> (d_hash, d_password, max_len);

  // Copy cracked password back
  HANDLE_ERROR (hipMemcpy (password, d_password, max_len, hipMemcpyDeviceToHost));
 
  // Free Device memory
  HANDLE_ERROR (hipFree (d_hash));
  HANDLE_ERROR (hipFree (d_password));
}



void sha1_init(sha1nfo *s) {
  memcpy(s->state.b,sha1InitState,HASH_LENGTH);
  s->byteCount = 0;
  s->bufferOffset = 0;
}

uint32_t sha1_rol32(uint32_t number, uint8_t bits) {
  return ((number << bits) | (number >> (32-bits)));
}

void sha1_hashBlock(sha1nfo *s) {
  uint8_t i;
  uint32_t a,b,c,d,e,t;

  a=s->state.w[0];
  b=s->state.w[1];
  c=s->state.w[2];
  d=s->state.w[3];
  e=s->state.w[4];
  for (i=0; i<80; i++) {
    if (i>=16) {
      t = s->buffer.w[(i+13)&15] ^ s->buffer.w[(i+8)&15] ^ s->buffer.w[(i+2)&15] ^ s->buffer.w[i&15];
      s->buffer.w[i&15] = sha1_rol32(t,1);
    }
    if (i<20) {
      t = (d ^ (b & (c ^ d))) + SHA1_K0;
    } else if (i<40) {
      t = (b ^ c ^ d) + SHA1_K20;
    } else if (i<60) {
      t = ((b & c) | (d & (b | c))) + SHA1_K40;
    } else {
      t = (b ^ c ^ d) + SHA1_K60;
    }
    t+=sha1_rol32(a,5) + e + s->buffer.w[i&15];
    e=d;
    d=c;
    c=sha1_rol32(b,30);
    b=a;
    a=t;
  }
  s->state.w[0] += a;
  s->state.w[1] += b;
  s->state.w[2] += c;
  s->state.w[3] += d;
  s->state.w[4] += e;
}

void sha1_addUncounted(sha1nfo *s, uint8_t data) {
  s->buffer.b[s->bufferOffset ^ 3] = data;
  s->bufferOffset++;
  if (s->bufferOffset == BLOCK_LENGTH) {
    sha1_hashBlock(s);
    s->bufferOffset = 0;
  }
}

void sha1_writebyte(sha1nfo *s, uint8_t data) {
  ++s->byteCount;
  sha1_addUncounted(s, data);
}

void sha1_write(sha1nfo *s, const char *data, size_t len) {
	for (;len--;) sha1_writebyte(s, (uint8_t) *data++);
}

void sha1_pad(sha1nfo *s) {
  // Implement SHA-1 padding (fips180-2 §5.1.1)

  // Pad with 0x80 followed by 0x00 until the end of the block
  sha1_addUncounted(s, 0x80);
  while (s->bufferOffset != 56) sha1_addUncounted(s, 0x00);

  // Append length in the last 8 bytes
  sha1_addUncounted(s, 0); // We're only using 32 bit lengths
  sha1_addUncounted(s, 0); // But SHA-1 supports 64 bit lengths
  sha1_addUncounted(s, 0); // So zero pad the top bits
  sha1_addUncounted(s, s->byteCount >> 29); // Shifting to multiply by 8
  sha1_addUncounted(s, s->byteCount >> 21); // as SHA-1 supports bitstreams as well as
  sha1_addUncounted(s, s->byteCount >> 13); // byte.
  sha1_addUncounted(s, s->byteCount >> 5);
  sha1_addUncounted(s, s->byteCount << 3);
}

uint8_t* sha1_result(sha1nfo *s) {
  int i;
  // Pad to complete the last block
  sha1_pad(s);
  
  // Swap byte order back
  for (i=0; i<5; i++) {
    uint32_t a,b;
    a=s->state.w[i];
    b=a<<24;
    b|=(a<<8) & 0x00ff0000;
    b|=(a>>8) & 0x0000ff00;
    b|=a>>24;
    s->state.w[i]=b;
  }
  
  // Return pointer to hash (20 characters)
  return s->state.b;
}

void printHash(uint8_t* hash) {
  int i;
  for (i=0; i<20; i++) {
    printf("%02x", hash[i]);
  }
  printf("\n");
}
